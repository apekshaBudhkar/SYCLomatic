// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Device/api_test32_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Device/api_test32_out/MainSourceFiles.yaml | wc -l > %T/Device/api_test32_out/count.txt
// RUN: FileCheck --input-file %T/Device/api_test32_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Device/api_test32_out


#include <hip/hip_runtime.h>
#include <nccl.h>

// CHECK: 4
// TEST_FEATURE: Device_get_device_id
int main() {
  int  device;
  ncclComm_t comm;
}