// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2
// RUN: dpct --format-range=none --use-custom-helper=api -out-root %T/LapackUtils/api_test9_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/LapackUtils/api_test9_out/MainSourceFiles.yaml | wc -l > %T/LapackUtils/api_test9_out/count.txt
// RUN: FileCheck --input-file %T/LapackUtils/api_test9_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/LapackUtils/api_test9_out

// CHECK: 31
// TEST_FEATURE: LapackUtils_geqrf


#include <hip/hip_runtime.h>
#include "hipsolver.h"

int main() {
  float* a_s;
  int64_t* tau_s;
  hipsolverHandle_t handle;
  size_t device_ws_size_s;
  size_t host_ws_size_s;
  hipsolverDnParams_t params;
  void* device_ws_s;
  void* host_ws_s;
  int *info;

  hipsolverDnXgeqrf(handle, params, 2, 2, HIP_R_32F, a_s, 2, HIP_R_32F, tau_s, HIP_R_32F, device_ws_s, device_ws_size_s, host_ws_s, host_ws_size_s, info);
  return 0;
}
